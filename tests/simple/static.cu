#include "hip/hip_runtime.h"
/*
 * CUDArrays is a library for easy multi-GPU program development.
 *
 * The MIT License (MIT)
 *
 * Copyright (c) 2013-2015 Barcelona Supercomputing Center and
 *                         University of Illinois
 *
 *  Developed by: Javier Cabezas <javier.cabezas@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE. */

#include <cstdio>

#include <cudarrays/static_array.hpp>

using namespace cudarrays;

template <typename T>
__host__ __device__
void test_single(T &A)
{
    for (unsigned i = 0; i < A.template dim<0>(); ++i) {
        for (unsigned j = 0; j < A.template dim<1>(); ++j) {
            A(i, j) = 1;
        }
    }

    A(1, 1) = 3;

    for (unsigned i = 0; i < A.template dim<0>(); ++i) {
        for (unsigned j = 0; j < A.template dim<1>(); ++j) {
            printf("%d ", A(i, j));
        }
        printf("\n");
    }
}

__global__
void test_kernel_single()
{
    static_array<int [3][3]> A;

    test_single(A);
}

static constexpr size_t BLOCK_X = 3;
static constexpr size_t BLOCK_Y = 3;

__global__
void test_kernel_shared()
{
    static_array<int [BLOCK_Y][BLOCK_X], memory_space::shared, layout::rmo, align<4>> A;

    bool first = threadIdx.y == 0 && threadIdx.x == 0;
    bool central = threadIdx.y == 1 && threadIdx.x == 1;

    int value = central? 3: 1;

    A(threadIdx.y, threadIdx.x) = value;

    __syncthreads();

    if (first)
        for (unsigned i = 0; i < A.dim<0>(); ++i) {
            for (unsigned j = 0; j < A.dim<1>(); ++j) {
                printf("%d ", A(i, j));
            }
            printf("\n");
        }
}

int main()
{
    static_array<int [3][3], memory_space::local, layout::rmo, align<1024, 2>> A;

    printf("Host\n");
    test_single(A);

    printf("Device\n");
    test_kernel_single<<<1, 1>>>();
    hipDeviceSynchronize();

    printf("Device __shared__\n");
    test_kernel_shared<<<1, dim3(BLOCK_X, BLOCK_Y)>>>();
    hipDeviceSynchronize();

    return 0;
}
